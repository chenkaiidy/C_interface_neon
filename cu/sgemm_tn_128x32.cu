
#include <hip/hip_runtime.h>

extern "C" __global__ void sgemm_tn_128x32(
    float*       param_C,
    const float* param_A,
    const float* param_B,
    float param_alpha,
    float param_beta,
    int   param_flags,
    int   param_lda,
    int   param_ldb,
    int   param_ldc,
    int   param_m,
    int   param_n,
    int   param_k,
    int   param_ldaz,
    int   param_ldbz,
    int   param_ldcz,
    int   param_batch_loops
) {

    __shared__ float share[(128*16 +  0)*2 + 32*16*2 + 4];
    *param_C = share[0];

}

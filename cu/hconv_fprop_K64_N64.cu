
#include <hip/hip_runtime.h>

extern "C" __global__ void hconv_fprop_K64_N64(
    float*     param_Sum,
    unsigned short*       param_O,
    const unsigned short* param_I,
    const unsigned short* param_F,
    float param_alpha,
    float param_beta,
    int param_flags,
    int param_offset_K,
    int param_N,
    int param_K,
    int param_D,
    int param_H,
    int param_W,
    int param_WN,
    int param_HWN,
    int param_DHWN,
    int param_C,
    int param_CRST,
    int param_RST,
    int param_RS,
    int param_magic_RS,
    int param_shift_RS,
    int param_S,
    int param_magic_S,
    int param_shift_S,
    int param_pad_d,
    int param_pad_h,
    int param_pad_w,
    int param_str_d,
    int param_str_h,
    int param_str_w,
    int param_Q,
    int param_PQ,
    int param_QN,
    int param_PQN,
    int param_MPQN,
    int param_magic_Q,
    int param_shift_Q,
    int param_magic_PQ,
    int param_shift_PQ
) {

    __shared__ float share[ 64*8*2 +  64*8*2 + 8];
    *param_Sum = share[0];

}


#include <hip/hip_runtime.h>

extern "C" __global__ void sconv_updat_C128_K64(
    float*     param_Sum,
    float*     param_F,
    const float* param_I,
    const float* param_E,
    float param_alpha,
    float param_beta,
    int param_flags,
    int param_offset_K,
    int param_N,
    int param_K,
    int param_D,
    int param_H,
    int param_W,
    int param_WN,
    int param_HWN,
    int param_DHWN,
    int param_C,
    int param_CRST,
    int param_RST,
    int param_magic_RST,
    int param_shift_RST,
    int param_RS,
    int param_magic_RS,
    int param_shift_RS,
    int param_S,
    int param_magic_S,
    int param_shift_S,
    int param_pad_d,
    int param_pad_h,
    int param_pad_w,
    int param_str_d,
    int param_str_h,
    int param_str_w,
    int param_P,
    int param_Q,
    int param_PQ,
    int param_QN,
    int param_PQN,
    int param_MPQN,
    int param_magic_Q,
    int param_shift_Q,
    int param_magic_PQ,
    int param_shift_PQ,
    int param_part_P,
    int param_part_Q,
    int param_part_PQ
) {

    __shared__ float share[(128*16 + 32)*2 + ( 64*16 + 32)*2 + 8];
    *param_Sum = share[0];

}

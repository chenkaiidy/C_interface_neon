
#include <hip/hip_runtime.h>

extern "C" __global__ void spool_bprop_avg(
    float*       param_E,
    float*       param_B,
    const float* param_I,
    int param_mode,
    int param_N,
    int param_W,
    int param_H,
    int param_D,
    int param_C,
    int param_WN,
    int param_HWN,
    int param_DHWN,
    int param_magic_H,
    int param_shift_H,
    int param_pad_w,
    int param_pad_h,
    int param_pad_d,
    int param_pad_c,
    int param_str_w,
    int param_str_h,
    int param_str_d,
    int param_str_c,
    int param_magic_str_w,
    int param_shift_str_w,
    int param_magic_str_h,
    int param_shift_str_h,
    int param_magic_str_d,
    int param_shift_str_d,
    int param_magic_str_c,
    int param_shift_str_c,
    int param_S,
    int param_R,
    int param_T,
    int param_J,
    int param_RS,
    int param_RST,
    int param_JRST,
    int param_magic_S,
    int param_shift_S,
    int param_magic_RS,
    int param_shift_RS,
    int param_magic_RST,
    int param_shift_RST,
    int param_Q,
    int param_P,
    int param_M,
    int param_K,
    int param_QN,
    int param_PQN,
    int param_MPQN
) {

    *param_E = 0;

}

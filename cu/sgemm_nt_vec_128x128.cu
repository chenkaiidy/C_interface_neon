
#include <hip/hip_runtime.h>

extern "C" __global__ void sgemm_nt_vec_128x128(
    float*       param_C,
    const float* param_A,
    const float* param_B,
    float param_alpha,
    float param_beta,
    int   param_flags,
    int   param_lda,
    int   param_ldb,
    int   param_ldc,
    int   param_m,
    int   param_n,
    int   param_k,
    int   param_ldaz,
    int   param_ldbz,
    int   param_ldcz,
    int   param_batch_loops
) {

    __shared__ float share[128*8*2 + 128*8*2 + 4];
    *param_C = share[0];

}

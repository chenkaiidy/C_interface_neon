
#include <hip/hip_runtime.h>

extern "C" __global__ void hgemm_nt_vec_128x128(
    unsigned short*       param_C,
    const unsigned short* param_A,
    const unsigned short* param_B,
    float param_alpha,
    float param_beta,
    int   param_flags,
    int   param_lda,
    int   param_ldb,
    int   param_ldc,
    int   param_m,
    int   param_n,
    int   param_k,
    int   param_ldaz,
    int   param_ldbz,
    int   param_ldcz,
    int   param_batch_loops
) {

    __shared__ float share[128*8*2 + 128*8*2 + 4];
    *param_C = share[0];

}
